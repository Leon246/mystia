#include "hip/hip_runtime.h"
/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "gpu_settings.h"
#include "EAMForceGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

//! tabulated function coeffs is set
int IsSet = 0;

//! Texture for reading particle positions
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;
// texture<float4, 1, hipReadModeElementType> force_data_tex;

//! Texture for tabulated function coeffs
// coeff.x = spline[3]
// coeff.y = spline[4]
// coeff.z = spline[5]
// coeff.w = spline[6]
texture<float4, 1, hipReadModeElementType> frho_coeff;
texture<float4, 1, hipReadModeElementType> rhor_coeff;
texture<float4, 1, hipReadModeElementType> z2r_coeff;
//! Texture for tabulated function coeffs
// coeff.x = spline[0]
// coeff.y = spline[1]
// coeff.z = spline[2]
// texture<float4, 1, hipReadModeElementType> frho_coeff_;
// texture<float4, 1, hipReadModeElementType> rhor_coeff_;
// texture<float4, 1, hipReadModeElementType> z2r_coeff_;

#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))

//! Kernel for calculating lj forces
/*! This kernel is called to calculate the lennard-jones forces on all N particles

	\param force_data Device memory array to write calculated forces to
	\param pdata Particle data on the GPU to calculate forces on
	\param nlist Neigbhor list data on the GPU to use to calculate the forces
	\param d_coeffs Coefficients to the lennard jones force.
	\param coeff_width Width of the coefficient matrix
	\param r_cutsq Precalculated r_cut*r_cut, where r_cut is the radius beyond which forces are
		set to 0
	\param rcut6inv Precalculated 1/r_cut**6
	\param xplor_denom_inv Precalculated 1/xplor denominator
	\param r_on_sq Precalculated r_on*r_on (for xplor)
	\param box Box dimensions used to implement periodic boundary conditions
	
	\a coeffs is a pointer to a matrix in memory. \c coeffs[i*coeff_width+j].x is \a lj1 for the type pair \a i, \a j.
	Similarly, .y is the \a lj2 parameter. The values in d_coeffs are read into shared memory, so 
	\c coeff_width*coeff_width*sizeof(float2) bytes of extern shared memory must be allocated for the kernel call.
	
	Developer information:
	Each block will calculate the forces on a block of particles.
	Each thread will calculate the total force on one particle.
	The neighborlist is arranged in columns so that reads are fully coalesced when doing this.
*/
template<bool ulf_workaround> __global__ void gpu_compute_eam_embed_kernel(gpu_force_data_arrays force_data, gpu_pdata_arrays pdata, gpu_boxsize box, gpu_nlist_array nlist, 
                                                                           float r_cutsq, int nr, int nrho, float rdr, float rdrho)
{	
	// start by identifying which particle we are to handle
	unsigned int idx_local = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx_local >= pdata.local_num)
		return;
	
	unsigned int idx_global = idx_local + pdata.local_beg;
	
	// load in the length of the list (MEM_TRANSFER: 4 bytes)
	unsigned int n_neigh = nlist.n_neigh[idx_global];

	// read in the position of our particle. Texture reads of float4's are faster than global reads on compute 1.0 hardware
	// (MEM TRANSFER: 16 bytes)
	float4 pos = tex1Dfetch(pdata_pos_tex, idx_global);
	
	// initialize the force to 0
	float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
  float rho = 0.0f;
	float p;
	unsigned int m;
	float4 coeff = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

	// prefetch neighbor index
	unsigned int cur_neigh = 0;
	unsigned int next_neigh = nlist.list[idx_global];

	// loop over neighbors
	// on pre C1060 hardware, there is a bug that causes rare and random ULFs when simply looping over n_neigh
	// the workaround (activated via the template paramter) is to loop over nlist.height and put an if (i < n_neigh)
	// inside the loop
	int n_loop;
	if (ulf_workaround)
		n_loop = nlist.height;
	else
		n_loop = n_neigh;
	
	for (int neigh_idx = 0; neigh_idx < n_loop; neigh_idx++) {
		if (!ulf_workaround || neigh_idx < n_neigh) {

		  // read the current neighbor index (MEM TRANSFER: 4 bytes)
		  // prefetch the next value and set the current one
		  cur_neigh = next_neigh;
		  next_neigh = nlist.list[nlist.pitch*(neigh_idx+1) + idx_global];
  		
		  // get the neighbor's position (MEM TRANSFER: 16 bytes)
		  float4 neigh_pos = tex1Dfetch(pdata_pos_tex, cur_neigh);
  		
		  // calculate dr (with periodic boundary conditions) (FLOPS: 3)
		  float dx = pos.x - neigh_pos.x;
		  float dy = pos.y - neigh_pos.y;
		  float dz = pos.z - neigh_pos.z;
  			
		  // apply periodic boundary conditions: (FLOPS 12)
		  dx -= box.Lx * rintf(dx * box.Lxinv);
		  dy -= box.Ly * rintf(dy * box.Lyinv);
		  dz -= box.Lz * rintf(dz * box.Lzinv);
  			
		  // calculate r squard (FLOPS: 5)
		  float rsq = dx*dx + dy*dy + dz*dz;
  		
      if (rsq >= r_cutsq)
			  rsq = 0.0f;
	
		  p = sqrt(rsq)*rdr + 1.0f;
		  m = floor(p);
		  m = MIN(m,nr-1);
		  p -= m;
		  p = MIN(p,1.0f);
		  coeff = tex1Dfetch(rhor_coeff, m);
		  rho += ((coeff.x*p+coeff.y)*p+coeff.z)*p+coeff.w;
		}
	}

  // embedding energy to each atom
  p = rho*rdrho + 1.0f;
	m = floor(p);
	m = MAX(1,MIN(m,nrho-1));
	p -= m;
	p = MIN(p, 1.0f);
	coeff = tex1Dfetch(frho_coeff, m);
	float fp = ((3.0f*coeff.x*p+2.0f*coeff.y)*p+coeff.z)*rdrho;
	float phi = ((coeff.x*p+coeff.y)*p+coeff.z)*p+coeff.w;
  force.x = 0.0f;
  force.y = 0.0f;
  force.z = phi;
  force.w = fp;
  
	force_data.force[idx_local] = force;
}

template<bool ulf_workaround> __global__ void gpu_compute_eam_force_kernel(gpu_force_data_arrays force_data, gpu_pdata_arrays pdata, gpu_boxsize box, gpu_nlist_array nlist,  float r_cutsq, unsigned int nr, unsigned int nrho, float rdr, float rdrho)
{
	// start by identifying which particle we are to handle
	unsigned int idx_local = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx_local >= pdata.local_num)
		return;
	
	unsigned int idx_global = idx_local + pdata.local_beg;
	
	// load in the length of the list (MEM_TRANSFER: 4 bytes)
	unsigned int n_neigh = nlist.n_neigh[idx_global];

	// read in the position of our particle. Texture reads of float4's are faster than global reads on compute 1.0 hardware
	// (MEM TRANSFER: 16 bytes)
	float4 pos = tex1Dfetch(pdata_pos_tex, idx_global);
  // get fake force
	// float4 fforce = tex1Dfetch(force_data_tex, idx_global);
  float4 fforce = force_data.force[idx_global];
	
	// initialize the force to 0
	float4 coeff;
	float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	float virial = 0.0f;

	// prefetch neighbor index
	unsigned int cur_neigh = 0;
	unsigned int next_neigh = nlist.list[idx_global];

	// loop over neighbors
	// on pre C1060 hardware, there is a bug that causes rare and random ULFs when simply looping over n_neigh
	// the workaround (activated via the template paramter) is to loop over nlist.height and put an if (i < n_neigh)
	// inside the loop
	int n_loop;
	if (ulf_workaround)
		n_loop = nlist.height;
	else
		n_loop = n_neigh;
		
	for (int neigh_idx = 0; neigh_idx < n_loop; neigh_idx++)
		{
		if (!ulf_workaround || neigh_idx < n_neigh)
		{
		// read the current neighbor index (MEM TRANSFER: 4 bytes)
		// prefetch the next value and set the current one
		cur_neigh = next_neigh;
		next_neigh = nlist.list[nlist.pitch*(neigh_idx+1) + idx_global];
		
		// get the neighbor's position (MEM TRANSFER: 16 bytes)
		float4 neigh_pos = tex1Dfetch(pdata_pos_tex, cur_neigh);
    // get the neighbor's fake fp
		// float4 neigh_fforce = tex1Dfetch(force_data_tex, cur_neigh);
    float4 neigh_fforce = force_data.force[cur_neigh];
				
		// calculate dr (with periodic boundary conditions) (FLOPS: 3)
		float dx = pos.x - neigh_pos.x;
		float dy = pos.y - neigh_pos.y;
		float dz = pos.z - neigh_pos.z;
			
		// apply periodic boundary conditions: (FLOPS 12)
		dx -= box.Lx * rintf(dx * box.Lxinv);
		dy -= box.Ly * rintf(dy * box.Lyinv);
		dz -= box.Lz * rintf(dz * box.Lzinv);
			
		// calculate r squard (FLOPS: 5)
		float rsq = dx*dx + dy*dy + dz*dz;
		
		if (rsq > r_cutsq)
			rsq = 0.0f;
	
		float p = sqrt(rsq)*rdr + 1.0f;
		unsigned int m = floor(p);
		m = MIN(m,nr-1);
		p -= m;
		p = MIN(p,1.0f);
		coeff = tex1Dfetch(rhor_coeff, m);
		float rhoip = ((3.0f*coeff.x*p+2.0f*coeff.y)*p+coeff.z)*rdr;
//		coeff = tex1Dfetch(rhor_coeff, m);
		float rhojp = ((3.0f*coeff.x*p+2.0f*coeff.y)*p+coeff.z)*rdr;
		
		coeff = tex1Dfetch(z2r_coeff, m);
		float z2p = ((3.0f*coeff.x*p+2.0f*coeff.y)*p+coeff.z)*rdr;
		float z2 = ((coeff.x*p+coeff.y)*p+coeff.z)*p+coeff.w;
		
		float recip = 1.0f/sqrt(rsq);
		float phi = z2*recip;
		float phip = z2p*recip - phi*recip;
		// pos.w -> fp[i], neigh_pos.w -> fp[j]
		float psip = fforce.w*rhojp + neigh_fforce.w*rhoip + phip;
		
		float forcemag_divr = -psip*recip;
		
		// calculate the virial (FLOPS: 3)
		virial += float(1.0/6.0) * rsq * forcemag_divr;

		// add up the force vector components (FLOPS: 7)
		force.x += dx * forcemag_divr;
		force.y += dy * forcemag_divr;
		force.z += dz * forcemag_divr;
		force.w += phi;
		}
		}
	force.w += fforce.z;
	// potential energy per particle must be halved
	force.w *= 0.5f;
	// now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes)
	force_data.force[idx_local] = force;
	force_data.virial[idx_local] = virial;
	}

/*! \param force_data Force data on GPU to write forces to
	\param pdata Particle data on the GPU to perform the calculation on
	\param box Box dimensions (in GPU format) to use for periodic boundary conditions
	\param nlist Neighbor list stored on the gpu
	\param d_coeffs A \a coeff_width by \a coeff_width matrix of coefficients indexed by type
		pair i,j. The x-component is lj1 and the y-component is lj2.
	\param coeff_width Width of the \a d_coeffs matrix.
	\param opt More execution options bundled up in a strct
	
	\returns Any error code resulting from the kernel launch
	
	This is just a driver for calcLJForces_kernel, see the documentation for it for more information.
*/
hipError_t gpu_compute_eam_forces(const gpu_force_data_arrays& force_data, const gpu_pdata_arrays &pdata, const gpu_boxsize &box,
                                   const gpu_nlist_array &nlist, const eam_options& opt, const gpu_eam_spline &spline)
{
  // setup the grid to run the kernel
  dim3 grid( (int)ceil((double)pdata.local_num / (double)opt.block_size), 1, 1);
  dim3 threads(opt.block_size, 1, 1);

	// bind the texture
	pdata_pos_tex.normalized = false;
	pdata_pos_tex.filterMode = hipFilterModePoint;
	hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata.pos, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;

  if (IsSet == 0) {
    // bind spline arrays
	  error = hipBindTexture(0, frho_coeff,  spline.frho_spline,  (opt.nrho+1)*sizeof(float4));
	  if (error != hipSuccess) return error;
	  error = hipBindTexture(0, rhor_coeff,  spline.rhor_spline,  (opt.nr+1)*sizeof(float4));
	  if (error != hipSuccess) return error;
	  error = hipBindTexture(0, z2r_coeff,   spline.z2r_spline,   (opt.nr+1)*sizeof(float4));
	  if (error != hipSuccess) return error;
    IsSet = 1;
  }

	// run the kernel
	if (opt.ulf_workaround)	{
		gpu_compute_eam_embed_kernel<true ><<<grid, threads>>>(force_data, pdata, box, nlist, opt.r_cutsq, opt.nr, opt.nrho, opt.rdr, opt.rdrho);
		gpu_compute_eam_force_kernel<true ><<<grid, threads>>>(force_data, pdata, box, nlist, opt.r_cutsq, opt.nr, opt.nrho, opt.rdr, opt.rdrho);
	}	else {
		gpu_compute_eam_embed_kernel<false><<<grid, threads>>>(force_data, pdata, box, nlist, opt.r_cutsq, opt.nr, opt.nrho, opt.rdr, opt.rdrho);
		gpu_compute_eam_force_kernel<false><<<grid, threads>>>(force_data, pdata, box, nlist, opt.r_cutsq, opt.nr, opt.nrho, opt.rdr, opt.rdrho);
	}

	if (!g_gpu_error_checking) {
		return hipSuccess;
	}	else {
		hipDeviceSynchronize();
		return hipGetLastError();
	}
}

// vim:syntax=cpp
